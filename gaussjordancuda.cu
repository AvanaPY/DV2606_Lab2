#include "hip/hip_runtime.h"
/***************************************************************************
 *
 * GPU version of Gauss-Jordan row reduction
 * Written by
 *  Emil Karlström, DVAMI19h
 *  Samuel Jonsson, DVAMI19h
 *
 ***************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <assert.h>

/*  If you would like to profile the program
    rename this macro to __PROFILE__ by removing the "z" */
#define __PROFILE__
#define MAX_SIZE 4096

typedef double matrix[MAX_SIZE][MAX_SIZE];

int	N;		/* matrix size		*/
int	maxnum;		/* max number of element*/
char* Init;		/* matrix init type	*/
int	PRINT;		/* print switch		*/
int BLOCK_DIM_SZ; /*  Max dimensional size of block, 
                        i.e each block is 
                        MAX_BLOCK_SIZE * MAX_BLOCK_SIZE threads*/
matrix	A;		        /* matrix A		*/
double	b[MAX_SIZE];	/* vector b             */
double	y[MAX_SIZE];	/* vector y             */

/* Verifying that the computed matrix is correct */
int VERIFY;
matrix verify_A;
double verify_b[MAX_SIZE];
double verify_y[MAX_SIZE];

/* forward declarations */
void work(void);
void Init_Matrix(void);
void Print_Matrix(void);
void Init_Default(void);
void verify_result(void);
int Read_Options(int, char**);

int
main(int argc, char** argv)
{
    printf("Gauss Jordan GPU\n");

#ifdef __PROFILE__
    clock_t GLOBAL_START, timestart, timeend;
    GLOBAL_START = clock();
    timestart = clock();
    double d;

#endif

    Init_Default();		/* Init default values	*/
    Read_Options(argc, argv);	/* Read arguments	*/
    Init_Matrix();		/* Init the matrix	*/

#ifdef __PROFILE__
    timeend = clock();
    d = (double)(timeend - timestart) / CLOCKS_PER_SEC;
    printf("Init default: %.3f\n", d);
    timestart = clock();
#endif

    /* Prepare verification */
    if(VERIFY == 1)
    {
        memcpy(verify_A, A, sizeof(double) * MAX_SIZE * MAX_SIZE);
        memcpy(verify_b, b, sizeof(double) * MAX_SIZE);
        memcpy(verify_y, y, sizeof(double) * MAX_SIZE);
    }

#ifdef __PROFILE__
    timeend = clock();
    d = (double)(timeend - timestart) / CLOCKS_PER_SEC;
    printf("Verify default: %.3f\n", d);
    timestart = clock();
#endif

    work();
    hipDeviceSynchronize();

#ifdef  __PROFILE__
    timeend = clock();
    printf("Total seconds for work(): %f\n", (double)(timeend - timestart) / CLOCKS_PER_SEC);
    timestart = clock();
#endif
    
    if (PRINT == 1)
        Print_Matrix();

    if(VERIFY == 1)
        verify_result();
        
#ifdef __PROFILE__
    timeend = clock();
    d = (double)(timeend - timestart) / CLOCKS_PER_SEC;
    printf("Verify: %.3f\n", d);
    d = (double)(timeend - GLOBAL_START) / CLOCKS_PER_SEC;
    printf("Program global time: %.3f\n", d);
#endif
}

__global__ void kernel_normalise_row(double* cuda_A, double* cuda_B, double* cuda_Y, int N, int k)
{
    int index = k + 1 + blockIdx.x * blockDim.x + threadIdx.x;
    if(index < N)
        cuda_A[k * N + index] = cuda_A[k * N + index] / cuda_A[k * N + k];
}

__global__ void kernel_norm_pivot(double* cuda_A, double* cuda_B, double* cuda_Y, int N, int k)
{
    cuda_Y[k] = cuda_B[k] / cuda_A[k * N + k];
    cuda_A[k * N + k] = 1;
}

__global__ void kernel_elimination(double* cuda_A, double* cuda_B, double* cuda_Y, int N, int k)
{
    int x = k + 1 + blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(y >= N || x >= N)
        return;

    // Boundary guard
    if(y != k)
        cuda_A[y * N + x] -= cuda_A[y * N + k] * cuda_A[k * N + x];
}

__global__ void kernel_eval(double* cuda_A, double* cuda_B, double* cuda_Y, int N, int k)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= N)
        return

    if(index < k)
    {
        cuda_Y[index] -= cuda_A[index * N + k] * cuda_Y[k];
        cuda_A[index * N + k] = 0.0;
    }
    else if(k < index)
    {
        cuda_B[index] -= cuda_A[index * N + k] * cuda_Y[k];
        cuda_A[index * N + k] = 0.0;
    } 
}

void
work(void)
{
    /* Allocate and copy data to GPU */
#ifdef __PROFILE__
    printf("Profiling work():\n");
    clock_t start, end;
    double diff, tot = 0;
    start = clock();
#endif

    double *cuda_A, *cuda_B, *cuda_Y;
    hipMalloc((void**)&cuda_A, sizeof(double) * N * N);
    hipMalloc((void**)&cuda_B, sizeof(double) * N);
    hipMalloc((void**)&cuda_Y, sizeof(double) * N);

#ifdef __PROFILE__
    end = clock();
    diff = (double)(end - start) / CLOCKS_PER_SEC;
    tot += diff;
    printf("\tcudaMalloc:  %.3fs\n", diff);

    start = clock();
#endif

    for(int k = 0; k < N; k++)
        hipMemcpy(cuda_A + N * k, A[k], sizeof(double) * N, hipMemcpyHostToDevice);
    hipMemcpy(cuda_B, b, sizeof(double) * N, hipMemcpyHostToDevice);
    
#ifdef __PROFILE__
    end = clock();
    diff = (double)(end - start) / CLOCKS_PER_SEC;
    tot += diff;
    printf("\tcudaMemcpy:  %.3fs\n", diff);
#endif

    /* GJ elimination */
    int block_size = BLOCK_DIM_SZ * BLOCK_DIM_SZ;
    int BLOCKS = max(1, N / block_size);
    
    dim3 blockDims(
        BLOCK_DIM_SZ, 
        BLOCK_DIM_SZ
    );
    dim3 gridDims(
        (int)ceil((float)N/(float)blockDims.x),
        (int)ceil((float)N/(float)blockDims.y)
    );

#ifdef __PROFILE__
    start = clock();
#endif

    int k;
    for(k = 0; k < N; k++)
    {
        /* Normalize */
        kernel_normalise_row<<<BLOCKS, block_size>>>(cuda_A, cuda_B, cuda_Y, N, k);
        kernel_norm_pivot<<<1, 1>>>(cuda_A, cuda_B, cuda_Y, N, k);
        
        /* Standard elimination and gauss-jordan thingies at the same time oh my god */
        kernel_elimination<<<gridDims, blockDims>>>(cuda_A, cuda_B, cuda_Y, N, k);
        
        /* Y evaluation */
        kernel_eval<<<BLOCKS, block_size>>>(cuda_A, cuda_B, cuda_Y, N, k);
    }
    hipDeviceSynchronize();
    
#ifdef __PROFILE__
    end = clock();
    diff = (double)(end - start) / CLOCKS_PER_SEC;
    tot += diff;
    printf("\tGaussJordan: %.3fs\n", diff);

    start = clock();
#endif

    /* Copy from GPU to RAM */

    /* 
        Copying A and B back to the Host is optional. We are not necessarily interested in A and B, but only the vector Y.
        A's final state is predictable as it's just a matrix of 0s with a diagonal of 1s, and the result of B is not of interest.
        Skipping copying A and B improves performance slightly.
    */ 
    hipMemcpy(y, cuda_Y, sizeof(double) * N, hipMemcpyDeviceToHost);

#ifdef __PROFILE__
    end = clock();
    diff = (double)(end - start) / CLOCKS_PER_SEC;
    tot += diff;
    printf("\tcudaMemcpy:  %.3fs\n", diff);

    start = clock();
    hipError_t e = hipGetLastError();
    const char* e_s = hipGetErrorString(e);
    printf("\tTotal time:  %.3f\n\tExiting on error: %s\n", tot, e_s);
#endif
    /* Print if we got any cool cuda errors */

    /* Free GPU memory; cuda is freeeeeeeeee~~~~~~~~ */
    hipFree(cuda_A);
    hipFree(cuda_B);
    hipFree(cuda_Y);
}

void
Init_Matrix()
{
    int i, j;

    printf("\nsize       = %dx%d", N, N);
    printf("\nBlock size = <%d,%d>", BLOCK_DIM_SZ, BLOCK_DIM_SZ);
    printf("\nmaxnum     = %d", maxnum);
    printf("\nInit	   = %s", Init);
    printf("\nInitializing matrix...");

    if (strcmp(Init, "rand") == 0) {
        for (i = 0; i < N; i++) {
            for (j = 0; j < N; j++) {
                if (i == j) /* diagonal dominance */
                    A[i][j] = (double)(rand() % maxnum) + 5.0;
                else
                    A[i][j] = (double)(rand() % maxnum) + 1.0;
            }
        }
    }
    if (strcmp(Init, "fast") == 0) {
        for (i = 0; i < N; i++) {
            for (j = 0; j < N; j++) {
                if (i == j) /* diagonal dominance */
                    A[i][j] = 5.0;
                else
                    A[i][j] = 2.0;
            }
        }
    }

    /* Initialize vectors b and y */
    for (i = 0; i < N; i++) {
        b[i] = 2.0;
        y[i] = 1.0;
    }

    printf("done \n\n");
    if (PRINT == 1)
        Print_Matrix();
}

void
Print_Matrix()
{
    int i, j;

    printf("Matrix A:\n");
    for (i = 0; i < N; i++) {
        printf("[");
        for (j = 0; j < N; j++)
            printf(" %5.2f,", A[i][j]);
        printf("]\n");
    }
    
    printf("Vector b:\n[");
    for (j = 0; j < N; j++)
        printf(" %5.2f,", b[j]);
    printf("]\n");

    printf("Vector y:\n[");
    for (j = 0; j < N; j++)
        printf(" %5.2f,", y[j]);
    printf("]\n");
    printf("\n\n");
}

void
Init_Default()
{
    N = 2048;
    Init = "fast";
    maxnum = 15.0;
    PRINT = 0;
    VERIFY = 0;

    BLOCK_DIM_SZ = 32;
}

int
Read_Options(int argc, char** argv)
{
    char* prog;

    prog = *argv;
    while (++argv, --argc > 0)
        if (**argv == '-')
            switch (*++ * argv) {
            case 'n':
                --argc;
                N = atoi(*++argv);
                break;
            case 'h':
                printf("\nHELP: try sor -u \n\n");
                exit(0);
                break;
            case 'u':
                printf("\nUsage: gaussian [-n problemsize]\n");
                printf("           [-D] show default values \n");
                printf("           [-h] help \n");
                printf("           [-I init_type] fast/rand \n");
                printf("           [-m maxnum] max random no \n");
                printf("           [-P print_switch] 0/1 \n");
                exit(0);
                break;
            case 'D':
                printf("\nDefault:  n         = %d ", N);
                printf("\n          Init      = rand");
                printf("\n          maxnum    = 5 ");
                printf("\n          P         = 0 \n\n");
                exit(0);
                break;
            case 'I':
                --argc;
                Init = *++argv;
                break;
            case 'm':
                --argc;
                maxnum = atoi(*++argv);
                break;
            case 'P':
                --argc;
                PRINT = atoi(*++argv);
                break;
            case 't':
                --argc;
                BLOCK_DIM_SZ = atoi(*++argv);
                break;
            case 'v':
                --argc;
                VERIFY = atoi(*++argv);
                break;
            default:
                printf("%s: ignored option: -%s\n", prog, *argv);
                printf("HELP: try %s -u \n\n", prog);
                break;
            }
    return 0;
}

double _round_to_decimals(double value, int decimals)
{
    int fac = pow(10, decimals - 1);
    return round(value * decimals) / decimals;
}

void verify_result()
{
    printf("Verifying result...\n");
    /* Gaussian elimination algorithm, Algo 8.4 from Grama */
    int k, j, i;
    for (k = 0; k < N; k++) { /* Outer loop */
        for (j = k + 1; j < N; j++)
            verify_A[k][j] = verify_A[k][j] / verify_A[k][k]; /* Division step */
        verify_y[k] = verify_b[k] / verify_A[k][k];
        verify_A[k][k] = 1.0;
        for (i = k + 1; i < N; i++) {
            for (j = k + 1; j < N; j++)
                verify_A[i][j] = verify_A[i][j] - verify_A[i][k] * verify_A[k][j]; /* Elimination step */
            verify_b[i] = verify_b[i] - verify_A[i][k] * verify_y[k];
            verify_A[i][k] = 0.0;
        }
        for (i = 0; i < k; i++) {
            for (j = k + 1; j < N; j++)
                verify_A[i][j] = verify_A[i][j] - verify_A[i][k] * verify_A[k][j]; /* Additional Elimination for Gauss-Jordan */
            verify_y[i] = verify_y[i] - verify_A[i][k] * verify_y[k];
            verify_A[i][k] = 0.0;
        }
    }
    printf("\tComputed correct matrix.\n");

    /* Print original matrix */

    if(PRINT == 1)
    {
        printf("Matrix A:\n");
        for (i = 0; i < N; i++) {
            printf("[");
            for (j = 0; j < N; j++)
                printf(" %5.15f,", A[i][j]);
            printf("]\n");
        }

        printf("\n");
        for (i = 0; i < N; i++) {
            printf("[");
            for (j = 0; j < N; j++)
                printf(" %5.15f,", verify_A[i][j]);
            printf("]\n");
        }
        
        printf("bs:\n[");
        for (j = 0; j < N; j++)
            printf(" %5.15f,", b[j]);
        printf("]\n[");
        for (j = 0; j < N; j++)
            printf(" %5.15f,", verify_b[j]);
        printf("]\n");

        printf("ys:\n[");
        for (j = 0; j < N; j++)
            printf(" %5.15f,", y[j]);
        printf("]\n[");
        for (j = 0; j < N; j++)
            printf(" %5.15f,", verify_y[j]);
        printf("]\n");
        printf("\n\n");
    }

    /* Print verify matrix */

    /* Verify they are still the same */
    int decimals = 10;
    for(int i = 0; i < N; i++)
    {
        /*  We do not need to verify the values of A and B
            because when we return from the CUDA we do not 
            copy the values as we are not interested in them
            however here is some code that would verify the results.
            Happy CUDA.*/
        // for(int j = 0; j < N; j++)
        //     assert(_round_to_decimals(verify_A[i][j], decimals) == _round_to_decimals(A[i][j], decimals) && "verify_A not equal to A");
        // assert(_round_to_decimals(verify_b[i], decimals) == _round_to_decimals(b[i], decimals) && "verify_b not equal to b");
        assert(_round_to_decimals(verify_y[i], decimals) == _round_to_decimals(y[i], decimals) && "verify_y not equal to y");
    }

    printf("\tPassed verification to %d decimals\n", decimals);
}