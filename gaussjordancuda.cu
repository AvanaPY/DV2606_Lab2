#include "hip/hip_runtime.h"
/***************************************************************************
 *
 * GPU version of Gauss-Jordan row reduction
 * Written by
 *  Emil Karlström, DVAMI19h
 *  Samuel Jonsson, DVAMI19h
 *
 ***************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define MAX_SIZE 4096
#define MAX_BLOCK_SIZE 1024

typedef double matrix[MAX_SIZE][MAX_SIZE];

int	N;		/* matrix size		*/
int	maxnum;		/* max number of element*/
char* Init;		/* matrix init type	*/
int	PRINT;		/* print switch		*/
matrix	A;		/* matrix A		*/
double	b[MAX_SIZE];	/* vector b             */
double	y[MAX_SIZE];	/* vector y             */

/* forward declarations */
void work(void);
void Init_Matrix(void);
void Print_Matrix(void);
void Init_Default(void);
int Read_Options(int, char**);

int
main(int argc, char** argv)
{
    printf("Gauss Jordan\n");
    // int i, timestart, timeend, iter;

    Init_Default();		/* Init default values	*/
    Read_Options(argc, argv);	/* Read arguments	*/
    Init_Matrix();		/* Init the matrix	*/
    work();
    if (PRINT == 1)
        Print_Matrix();

    hipDeviceSynchronize();
}

__global__ void kernel_normalize_row(double* cuda_A, double* cuda_B, double* cuda_Y, int N, int k)
{
    int index = k + 1 + blockIdx.x * blockDim.x + threadIdx.x;
    if(index < N)
    {
        cuda_A[k * N + index] = cuda_A[k * N + index] / cuda_A[k * N + k];
    }
}

__global__ void kernel_norm_pivot(double* cuda_A, double* cuda_B, double* cuda_Y, int N, int k)
{
    cuda_Y[k] = cuda_B[k] / cuda_A[k * N + k];
    cuda_A[k * N + k] = 1;
}

__global__ void kernel_elimination(double* cuda_A, double* cuda_B, double* cuda_Y, int N, int k)
{
    int index = k + 1 + blockIdx.x * blockDim.x + threadIdx.x;
    if(index < N)
    {
        int j;
        for(j = k + 1; j < N; j++)
        {
            cuda_A[index * N + j] -= cuda_A[index * N + k] * cuda_A[k * N + j];
        }
        cuda_B[index] -= cuda_A[index * N + k] * cuda_Y[k];
        cuda_A[index * N + k] = 0.0;
    }
}

__global__ void kernel_gj_step(double* cuda_A, double* cuda_B, double* cuda_Y, int N, int k)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int j;
    if(index < k)
    {
        for(j = k + 1; j < N; j++)
        {
            cuda_A[index * N + j] -= cuda_A[index * N + k] * cuda_A[k * N + j];
        }
        cuda_Y[index] -= cuda_A[index * N + k] * cuda_Y[k];
        cuda_A[index * N + k] = 0.0;
    }
}

void
work(void)
{
    /* Allocate and copy data to GPU */
    double *cuda_A, *cuda_B, *cuda_Y;
    hipMalloc((void**)&cuda_A, sizeof(double) * N * N);
    hipMalloc((void**)&cuda_B, sizeof(double) * N);
    hipMalloc((void**)&cuda_Y, sizeof(double) * N);

    for(int k = 0; k < N; k++)
        hipMemcpy(cuda_A + N * k, A[k], sizeof(double) * N, hipMemcpyHostToDevice);
    
    hipMemcpy(cuda_B, b, sizeof(double) * N, hipMemcpyHostToDevice);
    hipMemcpy(cuda_Y, y, sizeof(double) * N, hipMemcpyHostToDevice);

    /* GJ elimination */
    int BLOCKS = max(1, N / MAX_BLOCK_SIZE);
    int i,j,k;
    for(k = 0; k < N; k++)
    {
        /* Normalize */

        kernel_normalize_row<<<BLOCKS, MAX_BLOCK_SIZE>>>(cuda_A, cuda_B, cuda_Y, N, k);
        kernel_norm_pivot<<<1, 1>>>(cuda_A, cuda_B, cuda_Y, N, k);
        
        /* Standard elimination */
        kernel_elimination<<<BLOCKS, MAX_BLOCK_SIZE>>>(cuda_A, cuda_B, cuda_Y, N, k);
        
        /* Gauss Jordan step thingy*/
        kernel_gj_step<<<BLOCKS, MAX_BLOCK_SIZE>>>(cuda_A, cuda_B, cuda_Y, N, k);
    }

    /* Copy from GPU to RAM */

    for(int k = 0; k < N; k++)
        hipMemcpy(A[k], cuda_A + N * k, sizeof(double) * N, hipMemcpyDeviceToHost);
    hipMemcpy(b, cuda_B, sizeof(double) * N, hipMemcpyDeviceToHost);
    hipMemcpy(y, cuda_Y, sizeof(double) * N, hipMemcpyDeviceToHost);


    /* Print if we got any cool cuda errors */

    hipError_t e = hipGetLastError();
    const char* e_s = hipGetErrorString(e);

    /* Free GPU memory; cuda is freeeeeeeeee~~~~~~~~ */
    hipFree(cuda_A);
    hipFree(cuda_B);
    hipFree(cuda_Y);
}

void
Init_Matrix()
{
    int i, j;

    printf("\nsize      = %dx%d ", N, N);
    printf("\nmaxnum    = %d \n", maxnum);
    printf("Init	  = %s \n", Init);
    printf("Initializing matrix...");

    if (strcmp(Init, "rand") == 0) {
        for (i = 0; i < N; i++) {
            for (j = 0; j < N; j++) {
                if (i == j) /* diagonal dominance */
                    A[i][j] = (double)(rand() % maxnum) + 5.0;
                else
                    A[i][j] = (double)(rand() % maxnum) + 1.0;
            }
        }
    }
    if (strcmp(Init, "fast") == 0) {
        for (i = 0; i < N; i++) {
            for (j = 0; j < N; j++) {
                if (i == j) /* diagonal dominance */
                    A[i][j] = 5.0;
                else
                    A[i][j] = 2.0;
            }
        }
    }

    /* Initialize vectors b and y */
    for (i = 0; i < N; i++) {
        b[i] = 2.0;
        y[i] = 1.0;
    }

    printf("done \n\n");
    if (PRINT == 1)
        Print_Matrix();
}

void
Print_Matrix()
{
    int i, j;

    printf("Matrix A:\n");
    for (i = 0; i < N; i++) {
        printf("[");
        for (j = 0; j < N; j++)
            printf(" %5.2f,", A[i][j]);
        printf("]\n");
    }
    
    printf("Vector b:\n[");
    for (j = 0; j < N; j++)
        printf(" %5.2f,", b[j]);
    printf("]\n");

    printf("Vector y:\n[");
    for (j = 0; j < N; j++)
        printf(" %5.2f,", y[j]);
    printf("]\n");
    printf("\n\n");
}

void
Init_Default()
{
    N = 2048;
    Init = "fast";
    maxnum = 15.0;
    PRINT = 0;
}

int
Read_Options(int argc, char** argv)
{
    char* prog;

    prog = *argv;
    while (++argv, --argc > 0)
        if (**argv == '-')
            switch (*++ * argv) {
            case 'n':
                --argc;
                N = atoi(*++argv);
                break;
            case 'h':
                printf("\nHELP: try sor -u \n\n");
                exit(0);
                break;
            case 'u':
                printf("\nUsage: gaussian [-n problemsize]\n");
                printf("           [-D] show default values \n");
                printf("           [-h] help \n");
                printf("           [-I init_type] fast/rand \n");
                printf("           [-m maxnum] max random no \n");
                printf("           [-P print_switch] 0/1 \n");
                exit(0);
                break;
            case 'D':
                printf("\nDefault:  n         = %d ", N);
                printf("\n          Init      = rand");
                printf("\n          maxnum    = 5 ");
                printf("\n          P         = 0 \n\n");
                exit(0);
                break;
            case 'I':
                --argc;
                Init = *++argv;
                break;
            case 'm':
                --argc;
                maxnum = atoi(*++argv);
                break;
            case 'P':
                --argc;
                PRINT = atoi(*++argv);
                break;
            default:
                printf("%s: ignored option: -%s\n", prog, *argv);
                printf("HELP: try %s -u \n\n", prog);
                break;
            }
    return 0;
}