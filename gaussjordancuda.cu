#include "hip/hip_runtime.h"
/***************************************************************************
 *
 * GPU version of Gauss-Jordan row reduction
 * Written by
 *  Emil Karlström, DVAMI19h
 *  Samuel Jonsson, DVAMI19h
 *
 ***************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <assert.h>

#define MAX_SIZE 4096
int MAX_BLOCK_SIZE;

typedef double matrix[MAX_SIZE][MAX_SIZE];

int	N;		/* matrix size		*/
int	maxnum;		/* max number of element*/
char* Init;		/* matrix init type	*/
int	PRINT;		/* print switch		*/
matrix	A;		/* matrix A		*/
double	b[MAX_SIZE];	/* vector b             */
double	y[MAX_SIZE];	/* vector y             */

/* Verifying that the computed matrix is correct */
int VERIFY;
matrix verify_A;
double verify_b[MAX_SIZE];
double verify_y[MAX_SIZE];

/* forward declarations */
void work(void);
void Init_Matrix(void);
void Print_Matrix(void);
void Init_Default(void);
void verify_result(void);
int Read_Options(int, char**);

int
main(int argc, char** argv)
{
    printf("Gauss Jordan GPU\n");
    clock_t timestart, timeend;

    Init_Default();		/* Init default values	*/
    Read_Options(argc, argv);	/* Read arguments	*/
    Init_Matrix();		/* Init the matrix	*/

    /* Prepare verification */
    memcpy(verify_A, A, sizeof(double) * MAX_SIZE * MAX_SIZE);
    memcpy(verify_b, b, sizeof(double) * MAX_SIZE);
    memcpy(verify_y, y, sizeof(double) * MAX_SIZE);

    timestart = clock();
    work();
    hipDeviceSynchronize();
    timeend = clock();
    printf("Seconds used for computing: %f\n", (double)(timeend - timestart) / CLOCKS_PER_SEC);
    
    if (PRINT == 1)
        Print_Matrix();

    if(VERIFY == 1)
    {
        timestart = clock();
        verify_result();
        timeend = clock();
        printf("Seconds used for verification: %f\n", (double)(timeend - timestart) / CLOCKS_PER_SEC);
    }
}

__global__ void kernel_normalize_row(double* cuda_A, double* cuda_B, double* cuda_Y, int N, int k)
{
    int index = k + 1 + blockIdx.x * blockDim.x + threadIdx.x;
    if(index < N)
    {
        cuda_A[k * N + index] = cuda_A[k * N + index] / cuda_A[k * N + k];
    }
}

__global__ void kernel_norm_pivot(double* cuda_A, double* cuda_B, double* cuda_Y, int N, int k)
{
    cuda_Y[k] = cuda_B[k] / cuda_A[k * N + k];
    cuda_A[k * N + k] = 1;
}

__global__ void kernel_elimination(double* cuda_A, double* cuda_B, double* cuda_Y, int N, int k)
{
    int x = k + 1 + blockIdx.x * blockDim.x + threadIdx.x;
    int y = k + 1 + blockIdx.y * blockDim.y + threadIdx.y;

    // Boundary guard
    if((y < N) && (x < N))
        cuda_A[y * N + x] -= cuda_A[y * N + k] * cuda_A[k * N + x];
}

__global__ void kernel_eval_b(double* cuda_A, double* cuda_B, double* cuda_Y, int N, int k)
{
    int index = k + 1 + blockIdx.x * blockDim.x + threadIdx.x;
    if(index < N)
    {
        cuda_B[index] -= cuda_A[index * N + k] * cuda_Y[k];
        cuda_A[index * N + k] = 0.0;
    }
}
__global__ void kernel_gj_step(double* cuda_A, double* cuda_B, double* cuda_Y, int N, int k)
{
    int x = k + 1 + blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Boundary guard
    if((y < k) && (x < N))
        cuda_A[y * N + x] -= cuda_A[y * N + k] * cuda_A[k * N + x];
}
__global__ void kernel_gj_step2(double* cuda_A, double* cuda_B, double* cuda_Y, int N, int k)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < k)
    {
        cuda_Y[index] -= cuda_A[index * N + k] * cuda_Y[k];
        cuda_A[index * N + k] = 0.0;
    }
}

void
work(void)
{
    /* Allocate and copy data to GPU */
    double *cuda_A, *cuda_B, *cuda_Y;
    hipMalloc((void**)&cuda_A, sizeof(double) * N * N);
    hipMalloc((void**)&cuda_B, sizeof(double) * N);
    hipMalloc((void**)&cuda_Y, sizeof(double) * N);
    for(int k = 0; k < N; k++)
        hipMemcpy(cuda_A + N * k, A[k], sizeof(double) * N, hipMemcpyHostToDevice);
    
    hipMemcpy(cuda_B, b, sizeof(double) * N, hipMemcpyHostToDevice);
    hipMemcpy(cuda_Y, y, sizeof(double) * N, hipMemcpyHostToDevice);

    /* GJ elimination */
    int block_size = MAX_BLOCK_SIZE * MAX_BLOCK_SIZE;
    int BLOCKS = max(1, N / block_size);
    
    dim3 blockDims(
        MAX_BLOCK_SIZE, 
        MAX_BLOCK_SIZE
    );
    dim3 gridDims(
        (int)ceil((float)N/(float)blockDims.x),
        (int)ceil((float)N/(float)blockDims.y)
    );

    clock_t start, end;

    start = clock();
    int k;
    for(k = 0; k < N; k++)
    {
        /* Normalize */
        kernel_normalize_row<<<BLOCKS, block_size>>>(cuda_A, cuda_B, cuda_Y, N, k);
        kernel_norm_pivot<<<1, 1>>>(cuda_A, cuda_B, cuda_Y, N, k);
        
        /* Standard elimination */
        kernel_elimination<<<gridDims, blockDims>>>(cuda_A, cuda_B, cuda_Y, N, k);
        kernel_eval_b<<<BLOCKS, block_size>>>(cuda_A, cuda_B, cuda_Y, N, k);

        /* Gauss Jordan step thingy and zeroing numbers before*/
        kernel_gj_step<<<gridDims, blockDims>>>(cuda_A, cuda_B, cuda_Y, N, k);
        kernel_gj_step2<<<BLOCKS, block_size>>>(cuda_A, cuda_B, cuda_Y, N, k);
    }
    end = clock();
    double difference = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Total raw computing time: %f\n", difference);

    /* Copy from GPU to RAM */

    for(int k = 0; k < N; k++)
        hipMemcpy(A[k], cuda_A + N * k, sizeof(double) * N, hipMemcpyDeviceToHost);
    hipMemcpy(b, cuda_B, sizeof(double) * N, hipMemcpyDeviceToHost);
    hipMemcpy(y, cuda_Y, sizeof(double) * N, hipMemcpyDeviceToHost);

    /* Print if we got any cool cuda errors */

    hipError_t e = hipGetLastError();
    const char* e_s = hipGetErrorString(e);

    /* Free GPU memory; cuda is freeeeeeeeee~~~~~~~~ */
    hipFree(cuda_A);
    hipFree(cuda_B);
    hipFree(cuda_Y);
}

void
Init_Matrix()
{
    int i, j;

    printf("\nsize       = %dx%d", N, N);
    printf("\nBlock size = <%d,%d>", MAX_BLOCK_SIZE, MAX_BLOCK_SIZE);
    printf("\nmaxnum     = %d", maxnum);
    printf("\nInit	   = %s", Init);
    printf("\nInitializing matrix...");

    if (strcmp(Init, "rand") == 0) {
        for (i = 0; i < N; i++) {
            for (j = 0; j < N; j++) {
                if (i == j) /* diagonal dominance */
                    A[i][j] = (double)(rand() % maxnum) + 5.0;
                else
                    A[i][j] = (double)(rand() % maxnum) + 1.0;
            }
        }
    }
    if (strcmp(Init, "fast") == 0) {
        for (i = 0; i < N; i++) {
            for (j = 0; j < N; j++) {
                if (i == j) /* diagonal dominance */
                    A[i][j] = 5.0;
                else
                    A[i][j] = 2.0;
            }
        }
    }

    /* Initialize vectors b and y */
    for (i = 0; i < N; i++) {
        b[i] = 2.0;
        y[i] = 1.0;
    }

    printf("done \n\n");
    if (PRINT == 1)
        Print_Matrix();
}

void
Print_Matrix()
{
    int i, j;

    printf("Matrix A:\n");
    for (i = 0; i < N; i++) {
        printf("[");
        for (j = 0; j < N; j++)
            printf(" %5.2f,", A[i][j]);
        printf("]\n");
    }
    
    printf("Vector b:\n[");
    for (j = 0; j < N; j++)
        printf(" %5.2f,", b[j]);
    printf("]\n");

    printf("Vector y:\n[");
    for (j = 0; j < N; j++)
        printf(" %5.2f,", y[j]);
    printf("]\n");
    printf("\n\n");
}

void
Init_Default()
{
    N = 2048;
    Init = "fast";
    maxnum = 15.0;
    PRINT = 0;
    VERIFY = 0;

    MAX_BLOCK_SIZE = 32;
}

int
Read_Options(int argc, char** argv)
{
    char* prog;

    prog = *argv;
    while (++argv, --argc > 0)
        if (**argv == '-')
            switch (*++ * argv) {
            case 'n':
                --argc;
                N = atoi(*++argv);
                break;
            case 'h':
                printf("\nHELP: try sor -u \n\n");
                exit(0);
                break;
            case 'u':
                printf("\nUsage: gaussian [-n problemsize]\n");
                printf("           [-D] show default values \n");
                printf("           [-h] help \n");
                printf("           [-I init_type] fast/rand \n");
                printf("           [-m maxnum] max random no \n");
                printf("           [-P print_switch] 0/1 \n");
                exit(0);
                break;
            case 'D':
                printf("\nDefault:  n         = %d ", N);
                printf("\n          Init      = rand");
                printf("\n          maxnum    = 5 ");
                printf("\n          P         = 0 \n\n");
                exit(0);
                break;
            case 'I':
                --argc;
                Init = *++argv;
                break;
            case 'm':
                --argc;
                maxnum = atoi(*++argv);
                break;
            case 'P':
                --argc;
                PRINT = atoi(*++argv);
                break;
            case 't':
                --argc;
                MAX_BLOCK_SIZE = atoi(*++argv);
                break;
            case 'v':
                --argc;
                VERIFY = atoi(*++argv);
                break;
            default:
                printf("%s: ignored option: -%s\n", prog, *argv);
                printf("HELP: try %s -u \n\n", prog);
                break;
            }
    return 0;
}

double _round_to_decimals(double value, int decimals)
{
    int fac = pow(10, decimals - 1);
    return round((value * decimals) / decimals);
}

void verify_result()
{
    printf("Verifying result...\n");
    /* Gaussian elimination algorithm, Algo 8.4 from Grama */
    int k, j, i;
    for (k = 0; k < N; k++) { /* Outer loop */
        for (j = k + 1; j < N; j++)
            verify_A[k][j] = verify_A[k][j] / verify_A[k][k]; /* Division step */
        verify_y[k] = verify_b[k] / verify_A[k][k];
        verify_A[k][k] = 1.0;
        for (i = k + 1; i < N; i++) {
            for (j = k + 1; j < N; j++)
                verify_A[i][j] = verify_A[i][j] - verify_A[i][k] * verify_A[k][j]; /* Elimination step */
            verify_b[i] = verify_b[i] - verify_A[i][k] * verify_y[k];
            verify_A[i][k] = 0.0;
        }
        for (i = 0; i < k; i++) {
            for (j = k + 1; j < N; j++)
                verify_A[i][j] = verify_A[i][j] - verify_A[i][k] * verify_A[k][j]; /* Additional Elimination for Gauss-Jordan */
            verify_y[i] = verify_y[i] - verify_A[i][k] * verify_y[k];
            verify_A[i][k] = 0.0;
        }
    }
    printf("\tComputed correct matrix.\n");

    /* Print original matrix */

    if(PRINT == 1)
    {
        printf("Matrix A:\n");
        for (i = 0; i < N; i++) {
            printf("[");
            for (j = 0; j < N; j++)
                printf(" %5.15f,", A[i][j]);
            printf("]\n");
        }

        printf("\n");
        for (i = 0; i < N; i++) {
            printf("[");
            for (j = 0; j < N; j++)
                printf(" %5.15f,", verify_A[i][j]);
            printf("]\n");
        }
        
        printf("bs:\n[");
        for (j = 0; j < N; j++)
            printf(" %5.15f,", b[j]);
        printf("]\n[");
        for (j = 0; j < N; j++)
            printf(" %5.15f,", verify_b[j]);
        printf("]\n");

        printf("ys:\n[");
        for (j = 0; j < N; j++)
            printf(" %5.15f,", y[j]);
        printf("]\n[");
        for (j = 0; j < N; j++)
            printf(" %5.15f,", verify_y[j]);
        printf("]\n");
        printf("\n\n");
    }

    /* Print verify matrix */

    /* Verify they are still the same */
    int decimals = 10;
    for(int i = 0; i < N; i++)
    {
        for(int j = 0; j < N; j++)
            assert(_round_to_decimals(verify_A[i][j], decimals) == _round_to_decimals(A[i][j], decimals) && "verify_A not equal to A");
        assert(_round_to_decimals(verify_b[i], decimals) == _round_to_decimals(b[i], decimals) && "verify_b not equal to b");
        assert(_round_to_decimals(verify_y[i], decimals) == _round_to_decimals(y[i], decimals) && "verify_y not equal to y");
    }

    printf("\tPassed verification to %d decimals\n", decimals);
}